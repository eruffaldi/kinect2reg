#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>

#include <iostream>
#include "registrationK2.h"

/**
thrust::for_each(
    thrust::make_zip_iterator(
        thrust::make_tuple(map_dist.begin(), undistorted_data.begin(), map_x.begin(), map_yi.begin(),map_c_off.begin())),
    thrust::make_zip_iterator(
        thrust::make_tuple(map_dist.end(),   undistorted_data.end(),   map_x.end(),   map_yi.end(), map_c_off.end())),
        registerxy(...));

*/
struct registerxy
{
    __host__ __device__ 
    registerxy(const thrust::device_vector<float> & depth_data) :
        depth_data_(thrust::raw_pointer_cast(&depth_data[0]))
                {}

    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        const int index = thrust::get<0>(t);
        if(index < 0)
        {
            thrust::get<1>(t) = 0;
            thrust::get<4>(t) = -1;
        }
        else
        {
            const int size_color_ = 1920*1080;
            const float z = depth_data_[index];
            thrust::get<1>(t) = z; // undistorted
            const float rx = (thrust::get<2>(t) + (color_shift_m_ / z)) * color_fx_ + color_cx_;
            const int cx = rx;
            const int cy = thrust::get<3>(t); // map_yi
            const int c_off = cx + cy * 1920;
            if(c_off < 0 || c_off >= size_color_) {
                 thrust::get<4>(t) = -1;
            }
            else
            {
                 thrust::get<4>(t) = c_off; // in RGB data
            }
        }
    }

    float color_shift_m_;
    float color_fx_;
    float color_cx_;
    const float* depth_data_; // GPU
};

struct registerxy2
{
    __host__ __device__ 
    registerxy2(const thrust::device_vector<float> & depth_data) :
        depth_data_(thrust::raw_pointer_cast(&depth_data[0]))
                {}

    __host__ __device__ 
    registerxy2(const thrust::host_vector<float> & depth_data) :
        depth_data_(thrust::raw_pointer_cast(&depth_data[0]))
                {}

    __host__ __device__
    thrust::tuple<float,int> operator()(thrust::tuple<int,float,float> t)
    {
        const int index = thrust::get<0>(t);
        if(index < 0)
        {
            return thrust::make_tuple(0.0,-1);
        }
        else
        {
            const int size_color_ = 1920*1080;
            const float z = depth_data_[index];
            const float rx = (thrust::get<1>(t) + (color_shift_m_ / z)) * color_fx_ + color_cx_;
            const int cx = rx;
            const int cy = thrust::get<2>(t); // map_yi
            const int c_off = cx + cy * 1920;
            if(c_off < 0 || c_off >= size_color_) {
                 return thrust::make_tuple(z,-1);
            }
            else
            {
                return thrust::make_tuple(z,c_off);
            }
        }
    }

    float color_shift_m_;
    float color_fx_;
    float color_cx_;
    const float* depth_data_; // GPU
};

// TODO Super Filtering: given (cx,cy,z) write minimum z at (cx+-a,cy+-b)
// how is done on GPU....

// Example from ...
// https://github.com/code-iai/iai_kinect2/blob/master/kinect2_registration/src/depth_registration.cl
// in: idx = bufferIndex
//     zImg = bufferImgZ
//     dists = bufferDists filled by project
//     selDist = bufferSelDist filled by project
// in/out:    
//     rendered via idx = bufferRegistered[sizeRegistered] = 

/*
Without binning

fatomicMin(&(depthbuffer[dbindex].depthPrimTag),frag.depthPrimTag);

                            if(frag.depthPrimTag == depthbuffer[dbindex].depthPrimTag)//If this is true, we won the race condition
                                writeToDepthbuffer(x,y,frag, depthbuffer,resolution);

__device__ unsigned long long int fatomicMin(unsigned long long int  * addr, unsigned long long int value)
{
    unsigned long long ret = *addr;
    while(value < ret)
    {
        unsigned long long old = ret;
        if((ret = atomicCAS(addr, old, value)) == old)
            break;
    }
    return ret;

}                                
*/

int main(void)
{
    int w = 512;
    int h = 424;

    libfreenect2::Freenect2Device::IrCameraParams ips;
    libfreenect2::Freenect2Device::ColorCameraParams cps;
    libfreenect2::Registration rk2(ips,cps);

    // TODO init from C++ regular arrays
    thrust::device_vector<int> map_dist(w*h);//= rk2.distort_map;
    thrust::device_vector<int> map_x(w*h);//= depth_to_color_map_x;
    thrust::device_vector<int> map_yi(w*h);//= depth_to_color_map_yi; 

    thrust::device_vector<int> map_c_off(w*h); // output
    thrust::device_vector<int> undistorted_data(w*h); // output

    thrust::host_vector<float> tD(h*w); // depths in
    thrust::device_vector<float> depth_data = tD; // depth in GPU

    registerxy2 rxy(depth_data);
    rxy.color_shift_m_ = cps.shift_m;
    rxy.color_cx_ = cps.cx + 0.5;
    rxy.color_fx_ = cps.fx;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // transform form: return tuple
    thrust::transform(
        thrust::make_zip_iterator(thrust::make_tuple(map_dist.begin(), map_x.begin(), map_yi.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(map_dist.end(), map_x.end(), map_yi.end())),
        thrust::make_zip_iterator(thrust::make_tuple(undistorted_data.begin(), map_c_off.begin())),
        rxy);

    /*
    general form without return value
    
        thrust::for_each(
        thrust::make_zip_iterator(
            thrust::make_tuple(map_dist.begin(), undistorted_data.begin(), map_x.begin(), map_yi.begin(),map_c_off.begin())),
        thrust::make_zip_iterator(
            thrust::make_tuple(map_dist.end(),   undistorted_data.end(),   map_x.end(),   map_yi.end(), map_c_off.end())),
            rxy);
    */

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime; 
    hipEventElapsedTime(&elapsedTime , start, stop);
    printf("Avg. time is %f ms", elapsedTime/100);

    return 0;
}