#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>

#include <iostream>

#if 0
typedef thrust::tuple<float,float,float> Float3;

struct DotProduct : public thrust::binary_function<Float3,Float3,float>
{
    const float params[4];
    __host__ __device__
        float operator()(const Float3& a, const Float3& b) const
        {
            return thrust::get<0>(a) * thrust::get<0>(b) +    // x components
                   thrust::get<1>(a) * thrust::get<1>(b) +    // y components
                   thrust::get<2>(a) * thrust::get<2>(b);     // z components
        }
};
#endif

void preparemake3D(const float params[4], thrust::host_vector<float> & tW, thrust::host_vector<float> & tH,int w, int h)
{
    for(int i = 0; i < w; i++)
    {
        tW[i] = (i-params[1]+0.5)/params[0];
    }
    for (int i = 0; i < h; i++)
    {
        tH[i] = -(i-params[3]+0.5)/params[2];
    }
}

// x output = pw[c]*z
// y output = +-ph[c]*z
// z output = z
//
// xyz are coalesced and computed independently
// 1) replicate pw and ph across x and y
// 2) multiply by z


/*
                auto poc = (uint8_t*)(po+3);
                const float rx = pw[c]*z;
                const float ry = w*z;
                po[0] = rx;
                po[1] = -ry;
                po[2] = z;
                poc[0] = pc[2];
                poc[1] = pc[1];
                poc[2] = pc[0];
                good++;
                poi += 3*4+3;
*/

struct unprojectx
{
    __host__ __device__ 
    unprojectx(const thrust::device_vector<float> & p, int w) :p_(thrust::raw_pointer_cast(&p[0])), w_(w) {}
    
    __host__ __device__
    float operator()(const float & d, const int & c)
    {
        return d*p_[c % w_];
    }

    const float * p_;
    const int w_;
};

struct unprojecty
{
    __host__ __device__ 
    unprojecty(const thrust::device_vector<float> & p, int w) :p_(thrust::raw_pointer_cast(&p[0])), w_(w) {}

    __host__ __device__
    float operator()(const float & d, const int & c)
    {
        return d*p_[c / w_];
    }

    const float* p_;
    const int w_;
};

int main(void)
{
    float params[4] = {1,2,3,4}; // fx cx fy cy
    int w = 512;
    int h = 424;
    thrust::host_vector<float> tW(w);
    thrust::host_vector<float> tH(h);

    preparemake3D(params,tW,tH,w,h);

    // Copy host_vector H to device_vector D
    thrust::device_vector<float> dtW = tW,dtH = tH;

    thrust::host_vector<float> tD(h*w); // depths
    thrust::device_vector<float> dtD = tD;
    thrust::device_vector<float> dtPx(h*w); // points (all points)
    thrust::device_vector<float> dtPy(h*w); // points (all points)

    // works on: 1..w by 1..h 
    // input: tD depths
    // output expanded dtP
    // uses dtW and dtH as argument
    thrust::counting_iterator<int> co(0);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
        thrust::transform(dtD.begin(),dtD.end(), co, dtPx.begin(),unprojectx(tW,w)); 
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime; 
    hipEventElapsedTime(&elapsedTime , start, stop);
    printf("Avg. time is %f ms", elapsedTime/100);

    thrust::transform(dtD.begin(),dtD.end(),co,dtPy.begin(),unprojecty(tH,w));

    // filter by z


    return 0;
}